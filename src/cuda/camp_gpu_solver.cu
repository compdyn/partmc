#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Christian Guzman
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Interface Host-Device (CPU-GPU) to compute reaction-specific functions on GPU
 *
 */

//#include <hipsolver.h>
//#include <hip/hip_runtime.h>
//#include "camp_gpu_cusolver.h"

extern "C" {
#include "camp_gpu_solver.h"
#include "rxns_gpu.h"

// Reaction types (Must match parameters defined in pmc_rxn_factory)
#define RXN_ARRHENIUS 1
#define RXN_TROE 2
#define RXN_CMAQ_H2O2 3
#define RXN_CMAQ_OH_HNO3 4
#define RXN_PHOTOLYSIS 5
#define RXN_HL_PHASE_TRANSFER 6
#define RXN_AQUEOUS_EQUILIBRIUM 7
#define RXN_SIMPOL_PHASE_TRANSFER 10
#define RXN_CONDENSED_PHASE_ARRHENIUS 11
#define RXN_FIRST_ORDER_LOSS 12
#define RXN_EMISSION 13
#define RXN_WET_DEPOSITION 14

#define STREAM_RXN_ENV_GPU 0
#define STREAM_ENV_GPU 1
#define STREAM_DERIV_GPU 2

// Status codes for calls to camp_solver functions
#define CAMP_SOLVER_SUCCESS 0
#define CAMP_SOLVER_FAIL 1

//GPU async stream related variables to ensure robustness
//int n_solver_objects=0; //Number of solver_new_gpu calls
//hipStream_t *stream_gpu; //GPU streams to async computation/data movement
//int n_streams = 16;

//Gpu hardware info
//int model_data->max_n_gpu_thread;
//int model_data->max_n_gpu_blocks;

//Debug info
#ifdef PMC_DEBUG_GPU
  int counterDeriv = 0;       // Total calls to f()
  int counterJac = 0;         // Total calls to Jac()
  clock_t timeDeriv = 0;      // Compute time for calls to f()
  clock_t timeJac = 0;        // Compute time for calls to Jac()
  clock_t timeDerivKernel = 0; // Compute time for calls to f() kernel
  clock_t timeDerivSend = 0;
  clock_t timeDerivReceive = 0;
  clock_t timeDerivCPU = 0;
  clock_t t1 = 0;             //Auxiliar time counter
  clock_t t3 = 0;
#endif

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

/** \brief Allocate GPU solver variables
 *
 * \param n_dep_var number of solver variables per grid cell
 * \param n_state_var Number of variables on the state array per grid cell
 * \param n_rxn Number of reactions to include
 * \param n_rxn_int_param Total number of integer reaction parameters
 * \param n_rxn_float_param Total number of floating-point reaction parameters
 * \param n_cells Number of grid cells to solve simultaneously
 */
void solver_new_gpu_cu(ModelData *model_data, int n_dep_var,
                       int n_state_var, int n_rxn,
                       int n_rxn_int_param, int n_rxn_float_param, int n_rxn_env_param,
                       int n_cells) {
  //TODO: Select what % of data we want to compute on GPU simultaneously with CPU remaining %
  //Lengths
  model_data->state_size = n_state_var * n_cells * sizeof(double);
  model_data->deriv_size = n_dep_var * n_cells * sizeof(double);
  model_data->env_size = PMC_NUM_ENV_PARAM_ * n_cells * sizeof(double); //Temp and pressure
  model_data->rxn_env_data_size = n_rxn_env_param * n_cells * sizeof(double);
  model_data->rxn_env_data_idx_size = (n_rxn+1) * sizeof(int);
  //model_data->index_deriv_state_size = n_dep_var * n_cells * sizeof(int);
  model_data->small_data = 0;
  model_data->implemented_all = true;

  //TODO: cusolver
  //cusolver_test();

  //Allocate streams array and update variables related to streams
  //model_data->model_data_id = n_solver_objects;
  //if(n_solver_objects==0){
    //stream_gpu = (hipStream_t *)malloc(n_streams_limit * sizeof(hipStream_t));
      //model_data->stream_gpu = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
  //}
  //n_solver_objects++;

  //Alloc cpu
  //model_data->index_deriv_state = (int *)malloc(model_data->index_deriv_state_size);

  //Save positions to deriv in state
  /*int i_dep_var = 0;
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (model_data->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        model_data->index_deriv_state[i_dep_var] = i_spec + i_cell * n_state_var;//Save position
        i_dep_var++;
      }
    }
  }*/

  //Detect if we are working with few data values
  if (n_dep_var*n_cells < DATA_SIZE_LIMIT_OPT){
    model_data->small_data = 1;
  }

  //Set working GPU: we have 4 gpu available on power9. as default, it should be assign to gpu 0
  int device=0;
  hipSetDevice(device);

  //Set GPU properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  //Set max threads without triggering too many resources error
  model_data->max_n_gpu_thread = prop.maxThreadsPerBlock/2;
  model_data->max_n_gpu_blocks = prop.maxGridSize[1];
  int n_blocks = (n_rxn + model_data->max_n_gpu_thread - 1) / model_data->max_n_gpu_thread;

  //GPU allocation
  hipMalloc((void **) &model_data->deriv_gpu_data, model_data->deriv_size);
  hipMalloc((void **) &model_data->state_gpu, model_data->state_size);
  hipMalloc((void **) &model_data->env_gpu, model_data->env_size);
  hipMalloc((void **) &model_data->rxn_env_data_gpu, model_data->rxn_env_data_size);
  hipMalloc((void **) &model_data->rxn_env_data_idx_gpu, model_data->rxn_env_data_idx_size);
  //hipMalloc((void **) &model_data->index_deriv_state_gpu, model_data->index_deriv_state_size);

  //Setup GPU
  //HANDLE_ERROR(hipMemcpy(model_data->index_deriv_state_gpu, model_data->index_deriv_state, model_data->index_deriv_state_size, hipMemcpyHostToDevice));

  //GPU allocation few data on pinned memory
  if(model_data->small_data){
    //Notice auxiliar variables are created because we
    // can't pin directly variables initialized before
    hipHostMalloc((void**)&model_data->deriv_aux, model_data->deriv_size);
  }
  else{
    model_data->deriv_aux = (realtype *)malloc(model_data->deriv_size);
  }

  printf("small_data:%d\n", model_data->small_data);
  //printf("threads_per_block :%d\n", model_data->max_n_gpu_thread);

  //GPU create streams
  //for (int i = 0; i < n_streams; ++i)
  //  HANDLE_ERROR( hipStreamCreate(&model_data->stream_gpu[i]) );

  // Warning if exceeding GPU limits
  if( n_blocks > model_data->max_n_gpu_blocks){
    printf("\nWarning: More blocks assigned: %d than maximum block numbers: %d",
           n_blocks, model_data->max_n_gpu_blocks);
  }

#ifdef PMC_DEBUG_PRINT
  print_gpu_specs();
#endif

}

/** \brief Set reaction data on GPU prepared structure. RXN data is divided
 * into two different matrix, per double and int data respectively. Matrix are
 * reversed to improve memory access on GPU.
 *
 * \param md Pointer to the model data
 */

void solver_set_rxn_data_gpu(SolverData *sd) {

  ModelData *model_data = &(sd->model_data);
  int n_rxn = model_data->n_rxn;
  int n_cells = model_data->n_cells;
  unsigned int int_max_length = 0;
  unsigned int double_max_length = 0;

  //RXN lengths
  unsigned int int_lengths[n_rxn];
  unsigned int double_lengths[n_rxn];

  //Number of extra values added to square matrix(zeros and -1's)
  //unsigned int n_zeros[n_rxn];

  //Position on the matrix for each row
  unsigned int rxn_position[n_rxn];

  //Get lengths for int and double arrays
  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {

    // Set a WARNING if the reaction is not implemented yet on GPU
    bool implemented = false;
    int rxn_type = model_data->rxn_int_data[model_data->rxn_int_indices[i_rxn]];

    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM :
        implemented = false;
        break;
      case RXN_ARRHENIUS :
        implemented = true;
        break;
      case RXN_CMAQ_H2O2 :
        implemented = true;
        break;
      case RXN_CMAQ_OH_HNO3 :
        implemented = true;
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS :
        implemented = false;
        break;
      case RXN_EMISSION :
        implemented = true;
        break;
      case RXN_FIRST_ORDER_LOSS :
        implemented = true;
        break;
      case RXN_HL_PHASE_TRANSFER :
        implemented = false;
        break;
      case RXN_PHOTOLYSIS :
        implemented = true;
        break;
      case RXN_SIMPOL_PHASE_TRANSFER :
        implemented = false;
        break;
      case RXN_TROE :
        implemented = true;
        break;
      case RXN_WET_DEPOSITION :
        implemented = true;
        break;
    }
    if(!implemented){
#ifdef FAILURE_DETAIL
      printf("WARNING: Reaction type %d is not fully implemented on GPU. Computing on CPU...\n", rxn_type);
#endif
      model_data->implemented_all=false;
    }

    //Get RXN lengths
    int_lengths[i_rxn] = model_data->rxn_int_indices[i_rxn+1] - model_data->rxn_int_indices[i_rxn];
    double_lengths[i_rxn] = model_data->rxn_float_indices[i_rxn+1] - model_data->rxn_float_indices[i_rxn];

    //Update max size
    if(int_lengths[i_rxn]>int_max_length) int_max_length=int_lengths[i_rxn];
    if(double_lengths[i_rxn]>double_max_length) double_max_length=double_lengths[i_rxn];

    //Set initial position
    rxn_position[i_rxn] = i_rxn;

  }

  //Add a for to search the biggest distance int_max_length (ptrs[i] - ptrs[i-1]

  //Total lengths of rxn structure
  unsigned int rxn_int_length=n_rxn*int_max_length;
  unsigned int rxn_double_length=n_rxn*double_max_length;

  //Allocate int and double rxn data separately
  //Add -1 to avoid access and have a square matrix
  int *int_pointer = (int *) malloc(rxn_int_length * sizeof(int));
  memset(int_pointer, -1, rxn_int_length * sizeof(int));

  //Add 0 to avoid access and have a square matrix
  double *double_pointer = (double*)calloc(rxn_double_length, sizeof(double));

  //GPU allocation
  hipMalloc((void **) &model_data->int_pointer_gpu, rxn_int_length * sizeof(int));
  hipMalloc((void **) &model_data->double_pointer_gpu, rxn_double_length * sizeof(double));

  //Update number of zeros added on each reaction
  /*for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++)
    n_zeros[i_rxn] = (int_max_length - int_lengths[i_rxn]) +
                     (double_max_length - double_lengths[i_rxn]);*/

  //NOTE: no improvement on doing the sorting or not for gpu seems.
  //Sort by lengths
  //BubbleSort RXN by ascendant number of zeros for performance reasons
  //Fix reordered rxn give wrong values
  //bubble_sort_gpu(n_zeros, rxn_position, n_rxn);

  //Copy into gpu rxn data
  //Follows the rxn_position order
  //Rxn matrix is reversed to improve memory access on GPU
  //Matrix order is [int_length][n_rxn]

  int rxn_env_data_idx_aux[n_rxn];

  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {
    int i_pos=rxn_position[i_rxn];//i_rxn;//rxn_position[i_rxn];//for bubblesort
    for (int j = 0; j < int_lengths[i_pos]; j++){
      int *rxn_int_data = &(model_data->rxn_int_data[model_data->rxn_int_indices[i_pos]]);
      int_pointer[n_rxn*j + i_rxn] = rxn_int_data[j];
    }
    for (int j = 0; j < double_lengths[i_pos]; j++) {
      double *rxn_float_data = &(model_data->rxn_float_data[model_data->rxn_float_indices[i_pos]]);
      double_pointer[n_rxn*j + i_rxn] = rxn_float_data[j];
    }
    //Reorder the rate indices
    //Todo update on main code the rxn_env_data to read consecutively in cpu
    rxn_env_data_idx_aux[i_rxn] = model_data->rxn_env_idx[i_pos];
  }

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(model_data->int_pointer_gpu, int_pointer, rxn_int_length*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(model_data->double_pointer_gpu, double_pointer, rxn_double_length*sizeof(double), hipMemcpyHostToDevice));

  //Set rxn_env_data-idx
  HANDLE_ERROR(hipMemcpy(model_data->rxn_env_data_idx_gpu, rxn_env_data_idx_aux, model_data->rxn_env_data_idx_size, hipMemcpyHostToDevice));

  //Allocate jacobian
  model_data->jac_size = model_data->n_per_cell_solver_jac_elem * n_cells * sizeof(double);
  hipMalloc((void **) &model_data->jac_gpu_data, model_data->jac_size);

  if(model_data->small_data){
    hipHostMalloc((void**)&model_data->jac_aux, model_data->jac_size);
  }

  free(int_pointer);
  free(double_pointer);

}

void rxn_update_env_state_gpu(ModelData *model_data){

  // Get a pointer to the derivative data
  int n_cells = model_data->n_cells;
  int n_rxn = model_data->n_rxn;
  int n_threads = n_rxn*n_cells; //Reaction group per number of repetitions/cells
  double *state = model_data->total_state;
  double *rxn_env_data = model_data->rxn_env_data;
  double *env = model_data->total_env;
  int n_blocks = ((n_threads + model_data->max_n_gpu_thread - 1) / model_data->max_n_gpu_thread);

  //Faster, use for few values
  if (model_data->small_data){
    //This method of passing them as a function parameter has a theoric maximum of 4kb of data
    model_data->rxn_env_data_gpu= rxn_env_data;
    model_data->env_gpu= env;
  }
  //Slower, use for large values
  else{
/*
    //Async memcpy
    HANDLE_ERROR(hipMemcpyAsync(model_data->rxn_env_data_gpu, rxn_env_data,
            model_data->rxn_env_data_size, hipMemcpyHostToDevice, model_data->stream_gpu[STREAM_RXN_ENV_GPU]));
    HANDLE_ERROR(hipMemcpyAsync(model_data->env_gpu, env, model_data->env_size,
            hipMemcpyHostToDevice, model_data->stream_gpu[STREAM_ENV_GPU]));
*/
    HANDLE_ERROR(hipMemcpy(model_data->rxn_env_data_gpu, rxn_env_data,
                                 model_data->rxn_env_data_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(model_data->env_gpu, env, model_data->env_size,
                                 hipMemcpyHostToDevice));
  }

}

__global__
void camp_solver_update_model_state_cuda(double *total_state, double *y,
        int *index_deriv_state, double threshhold,double replacement_value, int *status)
{
  int i_dep_var = blockIdx.x * blockDim.x + threadIdx.x;
  if (y[i_dep_var] > -SMALL) {
    total_state[index_deriv_state[i_dep_var]] =
            y[i_dep_var] > threshhold ?
            y[i_dep_var] : replacement_value;
  } else {//error
    //*status = CAMP_SOLVER_FAIL;
  }
}

/*

//todo not working after first execution (I guess missiong free memory)
// and innefficient (increase the number of cudamemcpys)...
int camp_solver_update_model_state_gpu(N_Vector solver_state, ModelData *model_data,
        realtype threshhold, realtype replacement_value)
{
  int status = CAMP_SOLVER_SUCCESS; //CAMP_SOLVER_FAIL;
  int n_cells = model_data->n_cells;
  int n_state_var = model_data->n_per_cell_state_var;
  int n_dep_var = model_data->n_per_cell_dep_var;
  int n_threads = n_dep_var*n_cells;
  int n_blocks = ((n_threads + model_data->max_n_gpu_thread - 1) / model_data->max_n_gpu_thread);
  int *var_type = model_data->var_type;
  double *state = model_data->total_state;
  double *y = NV_DATA_S(solver_state);
  //int *index_deriv_state = model_data->index_deriv_state;

  //Need because f is also called in cvode first step initializations (cpu) :(
  HANDLE_ERROR(hipMemcpy(model_data->deriv_gpu_data, y, model_data->deriv_size, hipMemcpyHostToDevice));

  //Need because Jac (and maybe ohters) can also update total_model_state :(
  HANDLE_ERROR(hipMemcpy(model_data->state_gpu, model_data->total_state, model_data->state_size, hipMemcpyHostToDevice));

  camp_solver_update_model_state_cuda << < n_blocks, model_data->max_n_gpu_thread >> >
     (model_data->state_gpu, model_data->deriv_gpu_data, model_data->index_deriv_state_gpu,
     threshhold, replacement_value, &status);

  //Need because used in other cpu reactions (aero, sub_model, etc)
  HANDLE_ERROR(hipMemcpy(model_data->total_state, model_data->state_gpu, model_data->state_size, hipMemcpyDeviceToHost));

  //if failure detail and if error print the fail ala ya tu sabeh
  //Check error
  for(int i_dep_var = 0; i_dep_var < n_dep_var*n_cells; i_dep_var++)
  {
    if (NV_DATA_S(solver_state)[i_dep_var] < -SMALL) {
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update: [spec %d] = %le", i_spec,
                 NV_DATA_S(solver_state)[i_dep_var]);
#endif
      status = CAMP_SOLVER_FAIL;
      break;
    }
  }
  //printf("status %d\n",status);//why

  //status = CAMP_SOLVER_SUCCESS;
  //if nothing failed and we reach the end, then everything is correct
  //if (flag == CAMP_SOLVER_SUCCESS) status=flag;

  return status;
}

*/

/** \brief GPU function: Solve derivative
 *
 * \param state_init Pointer to first value of state array
 * \param deriv_init Pointer to first value of derivative array
 * \param time_step Current time step being computed (s)
 * \param deriv_length_cell Derivative length for one cell
 * \param model_data->state_size_cell Derivative length for one cell
 * \param n_rxn Number of reactions to include
 * \param n_cells_gpu Number of cells to compute
 * \param model_data->int_pointer Pointer to integer reaction data
 * \param model_data->double_pointer Pointer to double reaction data
 * \param rxn_env_data_init Pointer to first value of reaction rates
 */
__global__ void solveDerivative(double *state_init, double *deriv_init,
                                double time_step, int deriv_length_cell, int state_size_cell,
                                int rxn_env_data_size_cell, int n_rxn, int n_cells,
                                int *int_pointer, double *double_pointer,
                                double *rxn_env_data_init, int *rxn_env_data_idx,
                                double *env_init, int n_kernels, int i_kernel) //Interface CPU/GPU
{
  //Get thread id
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int offset1 = (n_rxn*n_cells/n_kernels)*i_kernel;
  int offset2 = (n_rxn*n_cells/n_kernels)*(i_kernel+1);

  //Maximum number of threads to compute all reactions
  if( (offset1 <= index) && (index < (offset2)) ){

    //Thread index for deriv and state,
    // till we don't finish all reactions of a cell, we stay on same index
    int i_cell=index/n_rxn;
    int i_rxn=index%n_rxn;

    //Another option: compute first the cells and then the reactions (seems working fine but no speedup)
    //Reorder rxn_env_data (first n_cells) to make this efficient (atm is worst for large n_cells)
    //int i_cell=index%n_cells;
    //int i_rxn=index/n_cells;

    //Get indices of each reaction
    double *rxn_float_data = (double *) &(((double *) double_pointer)[i_rxn]);
    int *int_data = (int *) &(((int *) int_pointer)[i_rxn]); //Same indices for each cell
    int rxn_type = int_data[0];
    int *rxn_int_data = (int *) &(int_data[1*n_rxn]);

    //Get indices for concentrations
    double *deriv_data = &( deriv_init[deriv_length_cell*i_cell]);
    double *state = &( state_init[state_size_cell*i_cell]);

    //Get indices for rates
    double *rxn_env_data = &(rxn_env_data_init
    [rxn_env_data_size_cell*i_cell+rxn_env_data_idx[i_rxn]]);

    //todo reduce model_data to allocate less memory per thread
    // (and trying to preserve matt cpu deriv code, maybe set different init parameters under COMPILE flag)
    ModelData model_data;
    model_data.grid_cell_state = &( state_init[state_size_cell*i_cell]);
    model_data.grid_cell_env = &( env_init[PMC_NUM_ENV_PARAM_*i_cell]);
    model_data.n_rxn = n_rxn;

    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM :
        rxn_gpu_aqueous_equilibrium_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                                       rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_ARRHENIUS :
        rxn_gpu_arrhenius_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_CMAQ_H2O2 :
        rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_CMAQ_OH_HNO3 :
        rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS :
        rxn_gpu_condensed_phase_arrhenius_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_EMISSION :
        rxn_gpu_emission_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_FIRST_ORDER_LOSS :
        rxn_gpu_first_order_loss_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_HL_PHASE_TRANSFER :
        //rxn_gpu_HL_phase_transfer_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
        //                                             rxn_float_data, rxn_env_data,time_stepn);
        break;
      case RXN_PHOTOLYSIS :
        rxn_gpu_photolysis_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER :
        //rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(md, rxn_env_data,
        //        state, deriv_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_TROE :
        rxn_gpu_troe_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                        rxn_float_data, rxn_env_data,time_step);
        break;
      case RXN_WET_DEPOSITION :
        rxn_gpu_wet_deposition_calc_deriv_contrib(&model_data, deriv_data, rxn_int_data,
                                             rxn_float_data, rxn_env_data,time_step);
        break;
    }
    __syncthreads();
  }

}

/** \brief Calculate the time derivative \f$f(t,y)\f$ on GPU
 *
 * \param model_data Pointer to the model data
 * \param deriv NVector to hold the calculated vector
 * \param time_step Current model time step (s)
 */
void rxn_calc_deriv_gpu(ModelData *model_data, N_Vector deriv, realtype time_step) {

  // Get a pointer to the derivative data
  realtype *deriv_data = N_VGetArrayPointer(deriv);
  int n_cells = model_data->n_cells;
  int n_kernels = 1; // Divide load into multiple kernel calls
  int n_rxn = model_data->n_rxn;
  int n_threads = n_rxn*n_cells; //Reaction group per number of repetitions/cells
  int n_blocks = ((n_threads + model_data->max_n_gpu_thread - 1) / model_data->max_n_gpu_thread);
  double *state = model_data->total_state;
  double *rxn_env_data = model_data->rxn_env_data;
  double *env = model_data->total_env;

#ifdef PMC_DEBUG_GPU
  t1 = clock();
#endif

/* //debug
  if(model_data->counterDeriv2==0){
    printf("camp solver_run start [(id),conc], n_state_var %d, n_cells %d\n", model_data->n_per_cell_state_var, n_cells);
    //printf("deriv_size %d\n", model_data->deriv_size);
    for (int i = 0; i < model_data->n_per_cell_state_var*n_cells; i++) {  // NV_LENGTH_S(deriv)
      printf("(%d) %-le \n",i+1, model_data->total_state[i]);
    }
  }
*/

  //Faster, use for few values
  if (model_data->small_data){
    //This method of passing them as a function parameter has a theoric maximum of 4kb of data
    model_data->state_gpu= state;
  }
    //Slower, use for large values
  else{
    HANDLE_ERROR(hipMemcpy(model_data->state_gpu, state, model_data->state_size, hipMemcpyHostToDevice));
  }

  //Reset deriv gpu
  HANDLE_ERROR(hipMemset(model_data->deriv_gpu_data, 0.0, model_data->deriv_size));

#ifdef PMC_DEBUG_GPU
  timeDerivSend += (clock() - t1);
  clock_t t2 = clock();
#endif

  //Loop to test multiple kernel executions
  for (int i_kernel=0; i_kernel<n_kernels;i_kernel++){
    //hipDeviceSynchronize();
    solveDerivative << < (n_blocks), model_data->max_n_gpu_thread >> >
     (model_data->state_gpu, model_data->deriv_gpu_data, time_step, model_data->n_per_cell_dep_var,
     model_data->n_per_cell_state_var, model_data->n_rxn_env_data,
     n_rxn, n_cells, model_data->int_pointer_gpu, model_data->double_pointer_gpu,
     model_data->rxn_env_data_gpu, model_data->rxn_env_data_idx_gpu, model_data->env_gpu,
     n_kernels, i_kernel);

  }

  hipDeviceSynchronize();

#ifdef PMC_DEBUG_GPU
  timeDerivKernel += (clock() - t2);
  t3 = clock();
#endif

  //Use pinned memory for few values
  if (model_data->small_data){
    HANDLE_ERROR(hipMemcpy(model_data->deriv_aux, model_data->deriv_gpu_data, model_data->deriv_size, hipMemcpyDeviceToHost));
    memcpy(deriv_data, model_data->deriv_aux, model_data->deriv_size);
  }
  else {
    //Async
    //HANDLE_ERROR(hipMemcpyAsync(model_data->deriv_aux, model_data->deriv_gpu_data,
    //model_data->deriv_size, hipMemcpyDeviceToHost, model_data->stream_gpu[STREAM_DERIV_GPU]));

    //Sync
    //HANDLE_ERROR(hipMemcpy(model_data->deriv_aux, model_data->deriv_gpu_data, model_data->deriv_size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(deriv_data, model_data->deriv_gpu_data, model_data->deriv_size, hipMemcpyDeviceToHost));
  }

  hipDeviceSynchronize();

/* //debug
  if(model_data->counterDeriv2==0){
    n_cells=2;
    for (int i = 0; i < n_cells; i++) {
      printf("cell %d \n", i);
      int size_j = NV_LENGTH_S(deriv) / n_cells;
      for (int j = 0; j < size_j; j++) {  // NV_LENGTH_S(deriv)
        printf("(%d) %-le \n", j + 1, NV_DATA_S(deriv)[j+i*size_j]);
      }
      printf("\n");
    }
  }
 */

#ifdef PMC_DEBUG_GPU
  timeDerivReceive += (clock() - t3);
  timeDeriv += (clock() - t1);
  t3 = clock();
#endif
}

/** \brief Fusion deriv data calculated from CPU and GPU
 * (either from funtions only implemented on CPU or work balancing between CPU and GPU)
 *
 * \param model_data Pointer to the model data
 * \param deriv NVector to hold the calculated vector
 * \param time_step Current model time step (s)
 */
void rxn_fusion_deriv_gpu(ModelData *model_data, N_Vector deriv) {

#ifdef PMC_DEBUG_GPU
 timeDerivCPU += (clock() - t3);
#endif
  // Get a pointer to the derivative data
  realtype *deriv_data = N_VGetArrayPointer(deriv);

  hipDeviceSynchronize();
  //HANDLE_ERROR(hipMemsetAsync(model_data->deriv_gpu_data, 0.0,
  //        model_data->deriv_size, model_data->stream_gpu[STREAM_DERIV_GPU]));

  if (model_data->small_data){
  }
  else {
    for (int i = 0; i < NV_LENGTH_S(deriv); i++) {  // NV_LENGTH_S(deriv)
      //Add to deriv the auxiliar contributions from gpu
      deriv_data[i] += model_data->deriv_aux[i];
    }
  }

}

#ifdef PMC_USE_SUNDIALS
void rxn_calc_deriv_cpu(ModelData *model_data, double *deriv_data,
                    realtype time_step) {

  //clock_t t = clock();

  // Get the number of reactions
  int n_rxn = model_data->n_rxn;

  // Loop through the reactions advancing the rxn_data pointer each time
  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {
    // Get pointers to the reaction data
    int *rxn_int_data =
        &(model_data->rxn_int_data[model_data->rxn_int_indices[i_rxn]]);
    double *rxn_float_data =
        &(model_data->rxn_float_data[model_data->rxn_float_indices[i_rxn]]);
    double *rxn_env_data =
        &(model_data->grid_cell_rxn_env_data[model_data->rxn_env_idx[i_rxn]]);

    // Get the reaction type
    int rxn_type = *(rxn_int_data++);

    // Call the appropriate function
    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM:
        rxn_gpu_aqueous_equilibrium_calc_deriv_contrib(model_data, deriv_data,
                                                   rxn_int_data, rxn_float_data,
                                                   rxn_env_data, time_step);
        break;
      case RXN_ARRHENIUS:
        rxn_gpu_arrhenius_calc_deriv_contrib(model_data, deriv_data, rxn_int_data,
                                         rxn_float_data, rxn_env_data,
                                         time_step);
        break;
      case RXN_CMAQ_H2O2:
        rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(model_data, deriv_data, rxn_int_data,
                                         rxn_float_data, rxn_env_data,
                                         time_step);
        break;
      case RXN_CMAQ_OH_HNO3:
        rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(model_data, deriv_data,
                                            rxn_int_data, rxn_float_data,
                                            rxn_env_data, time_step);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS:
        rxn_gpu_condensed_phase_arrhenius_calc_deriv_contrib(
            model_data, deriv_data, rxn_int_data, rxn_float_data, rxn_env_data,
            time_step);
        break;
      case RXN_EMISSION:
        rxn_gpu_emission_calc_deriv_contrib(model_data, deriv_data, rxn_int_data,
                                        rxn_float_data, rxn_env_data,
                                        time_step);
        break;
      case RXN_FIRST_ORDER_LOSS:
        rxn_gpu_first_order_loss_calc_deriv_contrib(model_data, deriv_data,
                                                rxn_int_data, rxn_float_data,
                                                rxn_env_data, time_step);
        break;
      case RXN_HL_PHASE_TRANSFER:
        //rxn_gpu_HL_phase_transfer_calc_deriv_contrib(model_data, deriv_data,
        //                                         rxn_int_data, rxn_float_data,
        //                                         rxn_env_data, time_step);
        break;
      case RXN_PHOTOLYSIS:
        rxn_gpu_photolysis_calc_deriv_contrib(model_data, deriv_data, rxn_int_data,
                                          rxn_float_data, rxn_env_data,
                                          time_step);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER:
        //rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(
        //   model_data, deriv_data, rxn_int_data, rxn_float_data, rxn_env_data,
        //    time_step);
        break;
      case RXN_TROE:
        rxn_gpu_troe_calc_deriv_contrib(model_data, deriv_data, rxn_int_data,
                                    rxn_float_data, rxn_env_data, time_step);
        break;
      case RXN_WET_DEPOSITION:
        rxn_gpu_wet_deposition_calc_deriv_contrib(model_data, deriv_data,
                                              rxn_int_data, rxn_float_data,
                                              rxn_env_data, time_step);
        break;
    }
  }

  //timeDeriv += (clock()- t);

}
#endif

/** \brief GPU function: Solve jacobian
 *
 * \param state_init Pointer to first value of state array
 * \param jac_init Pointer to first value of jacobian array
 * \param time_step Current time step being computed (s)
 * \param jac_length_cell jacobian length for one cell
 * \param model_data->state_size_cell jacobian length for one cell
 * \param n_rxn Number of reactions to include
 * \param n_cells_gpu Number of cells to compute
 * \param model_data->int_pointer Pointer to integer reaction data
 * \param model_data->double_pointer Pointer to double reaction data
 * \param rxn_env_data_init Pointer to first value of reaction rates
 */
__global__ void solveJacobian(double *state_init, double *jac_init,
                              double time_step, int jac_length_cell, int state_size_cell,
                              int rxn_env_data_size_cell, int n_rxn,
                              int n_cells, int *int_pointer, double *double_pointer,
                              double *rxn_env_data_init, int *rxn_env_data_idx) //Interface CPU/GPU
{
  //Get thread id
  /*int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Maximum number of threads to compute all reactions
  if(index < n_rxn*n_cells){

    //Thread index for jac and state,
    // till we don't finish all reactions of a cell, we stay on same index
    int i_cell=index/n_rxn;
    int i_rxn=index%n_rxn;

    //Get indices of each reaction
    int *int_data = (int *) &(((int *) int_pointer)[i_rxn]); //Same indices for each cell
    double *float_data = (double *) &(((double *) double_pointer)[i_rxn]);
    int rxn_type = int_data[0];
    int *rxn_int_data = (int *) &(int_data[1*n_rxn]);

    //Get indices for concentrations
    double *jac_data = &( jac_init[jac_length_cell*i_cell]);
    double *state = &( state_init[state_size_cell*i_cell]);

    //Get indices for rates
    double *rxn_env_data = &(rxn_env_data_init
    [rxn_env_data_size_cell*i_cell+rxn_env_data_idx[i_rxn]]);

    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM :
        //rxn_gpu_aqueous_equilibrium_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step, n_rxn);
        break;
      case RXN_ARRHENIUS :
        rxn_gpu_arrhenius_calc_jac_contrib(rxn_env_data,
                                           state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_CMAQ_H2O2 :
        rxn_gpu_CMAQ_H2O2_calc_jac_contrib(rxn_env_data,
                                           state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_CMAQ_OH_HNO3 :
        rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(rxn_env_data,
                                              state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS :
        //rxn_gpu_condensed_phase_arrhenius_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_EMISSION :
        rxn_gpu_emission_calc_jac_contrib(rxn_env_data,
                                          state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_FIRST_ORDER_LOSS :
        rxn_gpu_first_order_loss_calc_jac_contrib(rxn_env_data,
                                                  state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_HL_PHASE_TRANSFER :
        //rxn_gpu_HL_phase_transfer_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step, n_rxn);
        break;
      case RXN_PHOTOLYSIS :
        rxn_gpu_photolysis_calc_jac_contrib(rxn_env_data,
                                            state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER :
        //rxn_gpu_SIMPOL_phase_transfer_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_TROE :
        rxn_gpu_troe_calc_jac_contrib(rxn_env_data,
                                      state, jac_data, rxn_int_data, rxn_float_data, time_step, n_rxn);
        break;
      case RXN_WET_DEPOSITION :
        rxn_gpu_wet_deposition_calc_jac_contrib(rxn_env_data,
                                                state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
    }
    __syncthreads();
  }
   */

}


/** \brief Calculate the Jacobian on GPU
 *
 * \param model_data Pointer to the model data
 * \param J Jacobian to be calculated
 * \param time_step Current model time step (s)
 */

void rxn_calc_jac_gpu(SolverData *sd, SUNMatrix jac, realtype time_step) {

  //TODO: Fix jacobian with jac_ids...

  /*

  // Get a pointer to the jacobian data
  ModelData *model_data = &(sd->model_data);
  double *jac_data = SM_DATA_S(jac);
  int n_cells = model_data->n_cells;
  int n_rxn = model_data->n_rxn;
  int n_threads = n_rxn*n_cells; //Reaction group per number of repetitions/cells
  int n_blocks = ((n_threads + model_data->max_n_gpu_thread - 1) / model_data->max_n_gpu_thread);
  double *state = model_data->total_state;
  double *rxn_env_data = model_data->rxn_env_data;

  //Faster, use for few values
  if (model_data->small_data){
    //This method of passing them as a function parameter has a theoric maximum of 4kb of data
    model_data->state_gpu= state;
  }
    //Slower, use for large values
  else{
    HANDLE_ERROR(hipMemcpy(model_data->state_gpu, state, model_data->state_size, hipMemcpyHostToDevice));
  }

  HANDLE_ERROR(hipMemset(model_data->jac_gpu_data, 0, model_data->jac_size));

  solveJacobian << < (n_blocks), model_data->max_n_gpu_thread >> >
    (model_data->state_gpu, model_data->jac_gpu_data, time_step, model_data->n_per_cell_rxn_jac_elem,
    model_data->n_per_cell_state_var, model_data->n_rxn_env_data,
    n_rxn, n_cells, model_data->int_pointer_gpu, model_data->double_pointer_gpu, model_data->rxn_env_data_gpu, model_data->rxn_env_data_idx_gpu);

  hipDeviceSynchronize();// Secure cuda synchronization

  //Use pinned memory for few values
  if (model_data->small_data){
    HANDLE_ERROR(hipMemcpy(model_data->jac_aux, model_data->jac_gpu_data, model_data->jac_size, hipMemcpyDeviceToHost));
    memcpy(jac_data, model_data->jac_aux, model_data->jac_size);
  }
  else {
    HANDLE_ERROR(hipMemcpy(jac_data, model_data->jac_gpu_data, model_data->jac_size, hipMemcpyDeviceToHost));
  }

*/

}

/** \brief Free GPU data structures
 */
void free_gpu_cu(ModelData *model_data) {

#ifdef PMC_DEBUG_GPU
  printf("timeDeriv %lf\n", (((double)timeDeriv) ) / CLOCKS_PER_SEC); //*1000
  printf("timeDerivSend %lf\n", (((double)timeDerivSend) ) / CLOCKS_PER_SEC);
  printf("timeDerivKernel %lf\n", (((double)timeDerivKernel) ) / CLOCKS_PER_SEC);
  printf("timeDerivReceive %lf\n", (((double)timeDerivReceive) ) / CLOCKS_PER_SEC);
  printf("timeDerivCPU %lf\n", (((double)timeDerivCPU) ) / CLOCKS_PER_SEC);
#endif

  //for (int i = 0; i < n_streams; ++i)
  //  HANDLE_ERROR( hipStreamDestroy(model_data->stream_gpu[i]) );
/*

  */
  //free(model_data->jac_aux);
  HANDLE_ERROR(hipFree(model_data->int_pointer_gpu));
  HANDLE_ERROR(hipFree(model_data->double_pointer_gpu));
  HANDLE_ERROR(hipFree(model_data->deriv_gpu_data));
  //HANDLE_ERROR(hipFree(jac_gpu_data));

  if(model_data->small_data){
  }
  else{
    free(model_data->deriv_aux);
    HANDLE_ERROR(hipFree(model_data->state_gpu));
    HANDLE_ERROR(hipFree(model_data->env_gpu));
    HANDLE_ERROR(hipFree(model_data->rxn_env_data_gpu));
    HANDLE_ERROR(hipFree(model_data->rxn_env_data_idx_gpu));

  }

/*
  HANDLE_ERROR(hipFree(int_pointer_gpu));
  HANDLE_ERROR(hipFree(double_pointer_gpu));
  HANDLE_ERROR(hipFree(deriv_gpu_data));
  HANDLE_ERROR(hipFree(jac_gpu_data));

  if(small_data){
  }
  else{
    HANDLE_ERROR(hipFree(state_gpu));
    HANDLE_ERROR(hipFree(rxn_env_data_gpu));
    HANDLE_ERROR(hipFree(rxn_env_data_idx_gpu));
  }
*/
}

/* Auxiliar functions */

void bubble_sort_gpu(unsigned int *n_zeros, unsigned int *rxn_position, int n_rxn){

  int tmp,s=1,i_rxn=n_rxn;

  while(s){
    s=0;
    for (int i = 1; i < i_rxn; i++) {
      //Few zeros go first
      if (n_zeros[i] < n_zeros[i - 1]) {
        //Swap positions
        tmp = rxn_position[i];
        rxn_position[i] = rxn_position[i - 1];
        rxn_position[i - 1] = tmp;

        tmp = n_zeros[i];
        n_zeros[i] = n_zeros[i - 1];
        n_zeros[i - 1] = tmp;
        s=1;
      }
    }
    i_rxn--;
  }

}

/* Prints */

void print_gpu_specs() {

  printf("GPU specifications \n");

  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  maxGridSize: %d\n", prop.maxGridSize[1]);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  maxThreadsDim: %d\n", prop.maxThreadsDim[1]);
    printf("  totalGlobalMem: %zu\n", prop.totalGlobalMem);
    printf("  sharedMemPerBlock: %zu\n", prop.sharedMemPerBlock); //bytes
    printf("  multiProcessorCount: %d\n", prop.multiProcessorCount);
  }

}

// Old code (Not used now, but could be useful)
/*
 //use this instead of normal update_model_state? is less code
int camp_solver_update_model_state_cpu(N_Vector solver_state, ModelData *model_data,
                                       realtype threshhold, realtype replacement_value)
{
  int status = CAMP_SOLVER_FAIL;
  int n_cells = model_data->n_cells;
  int n_state_var = model_data->n_per_cell_state_var;
  int n_dep_var = model_data->n_per_cell_dep_var;
  int n_threads = n_state_var*n_cells;
  int n_blocks = ((n_threads + model_data->max_n_gpu_thread - 1) / model_data->max_n_gpu_thread);
  int *var_type = model_data->var_type;
  double *state = model_data->total_state;
  double *y = NV_DATA_S(solver_state);
  int *index_deriv_state = model_data->index_deriv_state;

  for(int i_dep_var = 0; i_dep_var < n_dep_var*n_cells; i_dep_var++)
  {
    if (NV_DATA_S(solver_state)[i_dep_var] > -SMALL) {
      model_data->total_state[index_deriv_state[i_dep_var]] =
              NV_DATA_S(solver_state)[i_dep_var] > threshhold
              ? NV_DATA_S(solver_state)[i_dep_var] : replacement_value;
      status = CAMP_SOLVER_SUCCESS;
    } else { //error
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update: [spec %d] = %le", i_spec,
                 NV_DATA_S(solver_state)[i_dep_var]);
#endif
      status = CAMP_SOLVER_FAIL;
      break;
    }
  }
  return status;
}
*/
}
