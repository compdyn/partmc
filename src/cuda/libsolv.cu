/* Copyright (C) 2020 Christian Guzman and Guillermo Oyarzun
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Basic GPU functions
 *
 */

#include<iostream>
#include<hip/hip_runtime.h>

#include<hip/hip_runtime_api.h>

//#include "libsolv2.h"

//#include<cublas.h> //todo fix cublas not compiling fine
//#include<cublas_v2.h>

using namespace std;

//
//dAthreads
//
// Para reservar memoria Double e Int
extern "C++" void cudaMallocDouble(double* &vector,int size)
{        
	hipMalloc((void**)&vector,size*sizeof(double));
}

extern "C++" void cudaMallocInt(int* &vector,int size)
{        
	hipMalloc((void**)&vector,size*sizeof(int));
}

// Para copiar a CPU->GPU Double e Int
extern "C++" void cudaMemcpyDToGpu(double* h_vect,double* d_vect,int size )
{
  hipMemcpy(d_vect,h_vect,size*sizeof(double),hipMemcpyHostToDevice);
}

extern "C++" void cudaMemcpyIToGpu(int* h_vect,int* d_vect,int size )
{
		hipMemcpy(d_vect,h_vect,size*sizeof(int),hipMemcpyHostToDevice);
}

// Para copiar a GPU->CPU Double e Int
extern "C++" void cudaMemcpyIToCpu(int* h_vect, int* d_vect,int size )
{
		hipMemcpy(h_vect,d_vect,size*sizeof(int),hipMemcpyDeviceToHost);
}

extern "C++" void cudaMemcpyDToCpu(double* h_vect, double* d_vect,int size )
{
  hipMemcpy(h_vect,d_vect,size*sizeof(double),hipMemcpyDeviceToHost);
}

// Para liberar memoria
extern "C++" void cudaFreeMem(void* vector)
{
	hipFree(vector);
}

extern "C++" void cudaGetLastErrorC(){
     hipError_t error;
     error=hipGetLastError();
     if(error!= hipSuccess)
     {
       cout<<" ERROR INSIDE A CUDA FUNCTION: "<<error<<" "<<hipGetErrorString(error)<<endl;
       exit(0);
     }
}

__global__ void cudamatScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      if(djA[j]==row)
      {
        dA[j] = 1.0 + alpha*dA[j];
      }
      else{
        dA[j] = alpha*dA[j];
      }
    }
  }
}

// A = I - gamma*J
// Based on CSR format, works on CSC too
// dA  : Matrix values (nnz size)
// djA : Matrix columns (nnz size)
// diA : Matrix rows (nrows+1 size)
// alpha : Scale factor
extern "C++" void gpu_matScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha, int blocks, int threads)
{

   blocks = (nrows+threads-1)/threads;
   
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

  cudamatScaleAddI<<<dimGrid,dimBlock>>>(nrows, dA, djA, diA, alpha);
}

// Diagonal precond
__global__ void cudadiagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    int jstart=diA[row];
    int jend  =diA[row+1];
    for(int j=jstart;j<jend;j++){
      if(djA[j]==row){
        if(dA[j]!=0.0)
          ddiag[row]= 1.0/dA[j];
        else{
          ddiag[row]= 1.0;
        }
      }
    }
  }

}

extern "C++" void gpu_diagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag, int blocks, int threads)
{

  blocks = (nrows+threads-1)/threads;

  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudadiagprecond<<<dimGrid,dimBlock>>>(nrows, dA, djA, diA, ddiag);
}

// y = constant
__global__ void cudasetconst(double* dy,double constant,int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=constant;
	}
}

extern "C++" void gpu_yequalsconst(double *dy, double constant, int nrows, int blocks, int threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 
   
   cudasetconst<<<dimGrid,dimBlock>>>(dy,constant,nrows);

}


// x=A*b
__global__ void cudaSpmvCSR(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    double sum = 0.0;
    for(int j=jstart; j<jend; j++)
    {
      sum+= db[djA[j]]*dA[j];
    }
    dx[row]=sum;
	}
 
}

__global__ void cudaSpmvCSC(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
	double mult;
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      mult = db[row]*dA[j];
      atomicAdd(&(dx[djA[j]]),mult);
    }
	}
}

extern "C++" void gpu_spmv(double* dx ,double* db, int nrows, double* dA, int *djA,int *diA,int mattype,int blocks,int  threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   if(mattype==0)
   {
     cudaSpmvCSR<<<dimGrid,dimBlock>>>(dx, db, nrows, dA, djA, diA);
   }
   else
   {
	    cudasetconst<<<dimGrid,dimBlock>>>(dx, 0.0, nrows);
	    cudaSpmvCSC<<<dimGrid,dimBlock>>>(dx, db, nrows, dA, djA, diA);
   }
}

// y= a*x+ b*y
__global__ void cudaaxpby(double* dy,double* dx, double a, double b, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]= a*dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_axpby(double* dy ,double* dx, double a, double b, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 
   
   cudaaxpby<<<dimGrid,dimBlock>>>(dy,dx,a,b,nrows);
}

// y = x
__global__ void cudayequalsx(double* dy,double* dx,int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=dx[row];
	}
}

extern "C++" void gpu_yequalsx(double *dy, double* dx, int nrows, int blocks, int threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 
   
   cudayequalsx<<<dimGrid,dimBlock>>>(dy,dx,nrows);

}

__global__ void cudadotxy(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;//*2 because init blocks is half
  //unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;//*2 because init blocks is half

  double mySum = (i < n) ? g_idata1[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  //for (unsigned int s=(blockDim.x+1)/2; s>0; s>>=1)
  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void cudareducey(double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;

  double mySum =  (tid < n) ? g_odata[tid] : 0;

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

//threads need to be pow of 2 //todo remove h_temp since not needed now
extern "C++" double gpu_dotxy(double* vec1, double* vec2, double* h_temp, double* d_temp, int nrows, int blocks,int threads)
{
  double sum;
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  //threads*sizeof(double)
  cudadotxy<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,nrows);
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);
  //printf("rho1 %f", sum);

  int redsize= sqrt(blocks) +1;
  redsize=pow(2,redsize);

  dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(redsize,1,1);

  cudareducey<<<dimGrid2,dimBlock2,redsize*sizeof(double)>>>(d_temp,blocks);
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);

  return sum;

/*
  cudaMemcpy(h_temp, d_temp, blocks * sizeof(double), cudaMemcpyDeviceToHost);
  double sum=0;
  for(int i=0;i<blocks;i++)
  {
    sum+=h_temp[i];
  }
  return sum;
*/
  /*dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(blocks,1,1);

  //Cuda only sum kernel call
  //cudareducey<<<dimGrid2,dimBlock2,blocks*sizeof(double)>>>(d_temp,blocks); //Takes quasi WAY MORE than cpu calc

  cudaMemcpy(h_temp, d_temp, sizeof(double), cudaMemcpyDeviceToHost);
  return h_temp[0];*/
}

/*
extern "C++" double gpu_dotxy(double *dy, double* dx, int nrows)
{
   double dot=0.0;
   cublasHandle_t hl;
   cublasCreate(&hl);

   cublasDdot(hl,nrows,dy,1,dx,1,&dot);

   cublasDestroy(hl);
   return dot;
}
*/

// z= a*z + x + b*y
__global__ void cudazaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=a*dz[row]  + dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_zaxpbypc(double* dz, double* dx ,double* dy, double a, double b, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 
   
   cudazaxpbypc<<<dimGrid,dimBlock>>>(dz,dx,dy,a,b,nrows);
}

// z= x*y
__global__ void cudamultxy(double* dz, double* dx,double* dy, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=dx[row]*dy[row];
	}
}

extern "C++" void gpu_multxy(double* dz, double* dx ,double* dy, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 
   
   cudamultxy<<<dimGrid,dimBlock>>>(dz,dx,dy,nrows);
}

// z= a*x + b*y
//__global__ void cudazaxpby(double* dz, double* dx,double* dy, double a, double b, int nrows)
__global__ void cudazaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=a*dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_zaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 

  cudazaxpby<<<dimGrid,dimBlock>>>(a,dx,b,dy,dz,nrows);
}

// y= a*x + y
__global__ void cudaaxpy(double* dy,double* dx, double a, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=a*dx[row] + dy[row];
	}
}

extern "C++" void gpu_axpy(double* dy, double* dx ,double a, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1); 
   
   cudaaxpy<<<dimGrid,dimBlock>>>(dy,dx,a,nrows);
}

// sqrt(sum ( (x_i*y_i)^2)/n)
__global__ void cudaDVWRMS_Norm(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  double mySum = (i < n) ? g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C++" double gpu_VWRMS_Norm(int n, double* vec1,double* vec2,double* h_temp,double* d_temp, int blocks,int threads)
{
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudaDVWRMS_Norm<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,n);

  //cudaMemcpy(h_temp, d_temp, blocks * sizeof(double), cudaMemcpyDeviceToHost);

  int redsize= sqrt(blocks) +1;
  redsize=pow(2,redsize);

  dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(redsize,1,1);

  cudareducey<<<dimGrid2,dimBlock2,redsize*sizeof(double)>>>(d_temp,blocks);

  double sum;
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);

  return sqrt(sum/n);

/*
  double sum=0;
  for(int i=0;i<blocks;i++)
  {
    sum+=h_temp[i];
  }
  return sqrt(sum/n);
  */
}

// y=alpha*y
__global__ void cudascaley(double* dy, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=a*dy[row];
  }
}

extern "C++" void gpu_scaley(double* dy, double a, int nrows, int blocks, int threads)
{
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudascaley<<<dimGrid,dimBlock>>>(dy,a,nrows);
}




// Device functions (equivalent to global functions but in device to allow calls from gpu)
__device__ void cudaDevicematScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      if(djA[j]==row)
      {
        dA[j] = 1.0 + alpha*dA[j];
      }
      else{
        dA[j] = alpha*dA[j];
      }
    }
  }
}

// Diagonal precond
__device__ void cudaDevicediagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    int jstart=diA[row];
    int jend  =diA[row+1];
    for(int j=jstart;j<jend;j++){
      if(djA[j]==row){
        if(dA[j]!=0.0)
          ddiag[row]= 1.0/dA[j];
        else{
          ddiag[row]= 1.0;
        }
      }
    }
  }

}

// y = constant
__device__ void cudaDevicesetconst(double* dy,double constant,int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=constant;
  }
}

// x=A*b
__device__ void cudaDeviceSpmvCSR(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    double sum = 0.0;
    for(int j=jstart; j<jend; j++)
    {
      sum+= db[djA[j]]*dA[j];
    }
    dx[row]=sum;
  }

}

__device__ void cudaDeviceSpmvCSC(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
  double mult;
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      mult = db[row]*dA[j];
      atomicAdd(&(dx[djA[j]]),mult);
//		dx[djA[j]]+= db[row]*dA[j];
    }
  }
}

// y= a*x+ b*y
__device__ void cudaDeviceaxpby(double* dy,double* dx, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]= a*dx[row] + b*dy[row];
  }
}

// y = x
__device__ void cudaDeviceyequalsx(double* dy,double* dx,int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=dx[row];
  }
}

__device__ void cudaDevicereducey(double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  //int id = blockIdx.x * blockDim.x + threadIdx.x;

  double mySum =  (tid < n) ? g_odata[tid] : 0;

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

//todo use mix of shared cuda and normal
__device__ void cudaDevicedotxy(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n, int n_shr_empty)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  //unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  //Used to ensure last block has 0 values for non-zero cases (Last block can have less cells than previous blocks)
  double mySum = (i < n) ? g_idata1[i]*g_idata2[i] : 0;

  //Init shr_memory to 0
  if(tid<blockDim.x/2)
    for (int j=0; j<2; j++)
      sdata[j*blockDim.x/2 + tid] = 0;

  //Set shr_memory to local values
  sdata[tid] = mySum;
  __syncthreads();

  //todo ensure that n_shr_empty is less than half of the max_threads to have enough threads
  //n_shr_empty its a different implementation from cuda reduce extended samples ( https://docs.nvidia.com/cuda/cuda-samples/index.html)
  // since n_threads_blocks isnotpowerof2
  // while these samples only takes into account n=notpowerof2, also we need active_threads able to be < max_threads
  // because other operations must work only with this number of threads to ensure work only with complete cells
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  //dont need to access global memory now
  //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
  *g_odata = sdata[0];
}

/*

//todo use mix of shared cuda and normal
__device__ void cudaDevicedotxy(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n, int n_shr_empty)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  //unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  //todo i<max_tid

  double mySum = (i < n) ? g_idata1[i]*g_idata2[i] : 0;

  //if (i + blockDim.x < n)
  //  mySum += g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x];//dont mix values from other blocks!

  //Last thread assign 0 to empty shr values
  if (tid == 0)//one thread
  {
    for (int j=0; j<n_shr_empty; j++)
      sdata[blockDim.x+j] = 0; //Assign 0 to non interesting sdata
  }
  sdata[tid] = mySum;
  __syncthreads();

  //todo ensure that n_shr_empty is less than half of the max_threads to have enough threads
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  //dont need to access global memory now
  //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
  *g_odata = sdata[0];
}

 */

// z= a*z + x + b*y
__device__ void cudaDevicezaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dz[row]=a*dz[row]  + dx[row] + b*dy[row];
  }
}

// z= x*y
__device__ void cudaDevicemultxy(double* dz, double* dx,double* dy, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dz[row]=dx[row]*dy[row];
  }
}

// z= a*x + b*y
__device__ void cudaDevicezaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dz[row]=a*dx[row] + b*dy[row];
  }
}

// y= a*x + y
__device__ void cudaDeviceaxpy(double* dy,double* dx, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=a*dx[row] + dy[row];
  }
}

// sqrt(sum ( (x_i*y_i)^2)/n)
__device__ void cudaDeviceDVWRMS_Norm(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  double mySum = (i < n) ? g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// y=alpha*y
__device__ void cudaDevicescaley(double* dy, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=a*dy[row];
  }
}

