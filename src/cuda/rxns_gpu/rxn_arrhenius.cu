#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Christian Guzman
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Arrhenius reaction solver functions
 *
*/
/** \file
 * \brief Arrhenius reaction solver functions
*/
extern "C"{
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../rxns_gpu.h"

#define TEMPERATURE_K_ env_data[0]
#define PRESSURE_PA_ env_data[1]

#define NUM_REACT_ int_data[0*n_rxn]
#define NUM_PROD_ int_data[1*n_rxn]
#define A_ float_data[0*n_rxn]
#define B_ float_data[1*n_rxn]
#define C_ float_data[2*n_rxn]
#define D_ float_data[3*n_rxn]
#define E_ float_data[4*n_rxn]
#define CONV_ float_data[5*n_rxn]
#define RATE_CONSTANT_ rxn_env_data[0*n_rxn]
#define NUM_INT_PROP_ 2
#define NUM_FLOAT_PROP_ 6
#define NUM_ENV_PARAM_ 1
#define REACT_(x) (int_data[(NUM_INT_PROP_ + x)*n_rxn]-1)
#define PROD_(x) (int_data[(NUM_INT_PROP_ + NUM_REACT_ + x)*n_rxn]-1)
#define DERIV_ID_(x) int_data[(NUM_INT_PROP_ + NUM_REACT_ + NUM_PROD_ + x)*n_rxn]
#define JAC_ID_(x) int_data[(NUM_INT_PROP_ + 2*(NUM_REACT_+NUM_PROD_) + x)*n_rxn]
#define YIELD_(x) float_data[(NUM_FLOAT_PROP_ + x)*n_rxn]
#define INT_DATA_SIZE_ (NUM_INT_PROP_+(NUM_REACT_+2)*(NUM_REACT_+NUM_PROD_))
#define FLOAT_DATA_SIZE_ (NUM_FLOAT_PROP_+NUM_PROD_)

/** \brief Flag Jacobian elements used by this reaction
 *
 * \param rxn_data A pointer to the reaction data
 * \param jac_struct 2D array of flags indicating potentially non-zero
 *                   Jacobian elements
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_arrhenius_get_used_jac_elem(void *rxn_data, bool **jac_struct)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  for (int i_ind = 0; i_ind < NUM_REACT_; i_ind++) {
    for (int i_dep = 0; i_dep < NUM_REACT_; i_dep++) {
      jac_struct[REACT_(i_dep)][REACT_(i_ind)] = true;
    }
    for (int i_dep = 0; i_dep < NUM_PROD_; i_dep++) {
      jac_struct[PROD_(i_dep)][REACT_(i_ind)] = true;
    }
  }

  return;
}

/** \brief Update the time derivative and Jacbobian array indices
 *
 * \param model_data Pointer to the model data
 * \param deriv_ids Id of each state variable in the derivative array
 * \param jac_ids Id of each state variable combo in the Jacobian array
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_arrhenius_update_ids(ModelData *model_data, int *deriv_ids,
                                    int **jac_ids, void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  // Update the time derivative ids
  for (int i=0; i < NUM_REACT_; i++)
    DERIV_ID_(i) = deriv_ids[REACT_(i)];
  for (int i=0; i < NUM_PROD_; i++)
    DERIV_ID_(i + NUM_REACT_) = deriv_ids[PROD_(i)];

  // Update the Jacobian ids
  int i_jac = 0;
  for (int i_ind = 0; i_ind < NUM_REACT_; i_ind++) {
    for (int i_dep = 0; i_dep < NUM_REACT_; i_dep++) {
      JAC_ID_(i_jac++) = jac_ids[REACT_(i_dep)][REACT_(i_ind)];
    }
    for (int i_dep = 0; i_dep < NUM_PROD_; i_dep++) {
      JAC_ID_(i_jac++) = jac_ids[PROD_(i_dep)][REACT_(i_ind)];
    }
  }
  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Do pre-derivative calculations
 *
 * Nothing to do for arrhenius reactions
 *
 * \param model_data Pointer to the model data, including the state array
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_arrhenius_pre_calc(ModelData *model_data, void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Calculate contributions to the time derivative \f$f(t,y)\f$ from
 * this reaction.
 *
 * \param model_data Pointer to the model data, including the state array
 * \param deriv Pointer to the time derivative to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being computed (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
#ifdef PMC_USE_SUNDIALS

#ifdef __CUDA_ARCH__//maybe is better if we activate gpu? because if we dont activate but cuda_Arch then PUM (well if not work a replace to pmc_use_gpu is easy)
__host__ __device__
#endif
void rxn_gpu_arrhenius_calc_deriv_contrib(ModelData *model_data, realtype *deriv,
                                      int *rxn_int_data, double *rxn_float_data,
                                      double *rxn_env_data, double time_step)
{
#ifdef __CUDA_ARCH__
  int n_rxn=model_data->n_rxn;
#else
  int n_rxn=1;
#endif
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double *env_data = model_data->grid_cell_env;

  double rate = RATE_CONSTANT_;
  for (int i_spec=0; i_spec<NUM_REACT_; i_spec++)
    rate *= state[REACT_(i_spec)];

  // Add contributions to the time derivative
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<NUM_REACT_; i_spec++, i_dep_var++) {
      if (DERIV_ID_(i_dep_var) < 0) continue;
#ifdef __CUDA_ARCH__
        atomicAdd(&(deriv[DERIV_ID_(i_dep_var)]),-rate);
        //atomicAdd(&(deriv[DERIV_ID_(i_dep_var)]),0.5); //debug
#else
        deriv[DERIV_ID_(i_dep_var)] -= rate;
#endif
	}
    for (int i_spec=0; i_spec<NUM_PROD_; i_spec++, i_dep_var++) {
      if (DERIV_ID_(i_dep_var) < 0) continue;

      // Negative yields are allowed, but prevented from causing negative
      // concentrations that lead to solver failures
      if (-rate*YIELD_(i_spec)*time_step <= state[PROD_(i_spec)]) {
#ifdef __CUDA_ARCH__
        atomicAdd(&(deriv[DERIV_ID_(i_dep_var)]),rate*YIELD_(i_spec));
        //atomicAdd(&(deriv[DERIV_ID_(i_dep_var)]),0.1); //debug
#else
        deriv[DERIV_ID_(i_dep_var)] += rate * YIELD_(i_spec);
#endif
      }
    }
  }

}



#endif

/** \brief Calculate contributions to the Jacobian from this reaction
 *
 * \param model_data Pointer to the model data
 * \param J Pointer to the sparse Jacobian matrix to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being calculated (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
#ifdef PMC_USE_SUNDIALS
#ifdef __CUDA_ARCH__
__host__ __device__
#endif
void rxn_gpu_arrhenius_calc_jac_contrib(ModelData *model_data, realtype *J, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step)
{
#ifdef __CUDA_ARCH__
  int n_rxn=model_data->n_rxn;
#else
  int n_rxn=1;;
#endif
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double *env_data = model_data->grid_cell_env;

  // Add contributions to the Jacobian
  int i_elem = 0;
  for (int i_ind = 0; i_ind < NUM_REACT_; i_ind++) {
    // Calculate d_rate / d_i_ind
    realtype rate = RATE_CONSTANT_;
    for (int i_spec = 0; i_spec < NUM_REACT_; i_spec++)
      if (i_spec != i_ind) rate *= state[REACT_(i_spec)];

    for (int i_dep=0; i_dep<NUM_REACT_; i_dep++, i_elem++) {
      if (JAC_ID_(i_elem) < 0) continue;
#ifdef __CUDA_ARCH__
      atomicAdd(&(J[JAC_ID_(i_elem)]),-rate);
#else
      J[JAC_ID_(i_elem)] -= rate;
#endif
    }
    for (int i_dep=0; i_dep<NUM_PROD_; i_dep++, i_elem++) {
  if (JAC_ID_(i_elem) < 0) continue;
      // Negative yields are allowed, but prevented from causing negative
      // concentrations that lead to solver failures
      if (-rate * state[REACT_(i_ind)] * YIELD_(i_dep) * time_step <= state[PROD_(i_dep)]) {
#ifdef __CUDA_ARCH__
    atomicAdd(&(J[JAC_ID_(i_elem)]),YIELD_(i_dep) * rate);
#else
    J[JAC_ID_(i_elem)] += YIELD_(i_dep) * rate;
#endif
      }
    }
  }

}
#endif

/** \brief Retrieve Int data size
 *
 * \param rxn_data Pointer to the reaction data
 * \return The data size of int array
 */
void * rxn_gpu_arrhenius_get_float_pointer(void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);


  return (void*) float_data;
}

/** \brief Advance the reaction data pointer to the next reaction
 *
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_arrhenius_skip(void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);


  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}

/** \brief Print the Arrhenius reaction parameters
 *
 * \param rxn_data Pointer to the reaction data
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
void * rxn_gpu_arrhenius_print(void *rxn_data)
{
  int n_rxn=1;
  int *int_data = (int*) rxn_data;
  double *float_data = (double*) &(int_data[INT_DATA_SIZE_]);

  printf("\n\nArrhenius reaction\n");
  for (int i=0; i<INT_DATA_SIZE_; i++)
    printf("  int param %d = %d\n", i, int_data[i]);
  for (int i=0; i<FLOAT_DATA_SIZE_; i++)
    printf("  float param %d = %le\n", i, float_data[i]);

  return (void*) &(float_data[FLOAT_DATA_SIZE_]);
}

}
